#include <hip/hip_runtime.h>
#include <iostream>

__global__ void cuGEMM(const float *A, const float *B, float *C, int N, int K, int M) {
    // Calculate row and column index of the element in the output matrix C
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < M) {
        float value = 0.0f;
        // Compute the dot product of the row of A and column of B
        for (int i = 0; i < K; ++i) {
            value += A[row * K + i] * B[i * M + col];
        }
        // Write the result to the output matrix C
        C[row * M + col] = value;
    }
}

void launchGEMM(const float *A, const float *B, float *C, int N, int K, int M) {
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N * K * sizeof(float));
    hipMalloc((void**)&d_B, K * M * sizeof(float));
    hipMalloc((void**)&d_C, N * M * sizeof(float));

    hipMemcpy(d_A, A, N * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, K * M * sizeof(float), hipMemcpyHostToDevice);

    // Define block and grid dimensions
    dim3 blockSize(16, 16); // 16x16 threads per block
    dim3 gridSize((M + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

    // Launch the kernel
    cuGEMM<<<gridSize, blockSize>>>(d_A, d_B, d_C, N, K, M);

    // Copy the result back to the host
    hipMemcpy(C, d_C, N * M * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
