#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vectorAdd(const float *A, const float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

void launchVectorAdd(const float *A, const float *B, float *C, int N) {
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N * sizeof(float));
    hipMalloc((void**)&d_B, N * sizeof(float));
    hipMalloc((void**)&d_C, N * sizeof(float));

    hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;
    vectorAdd<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);

    hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
